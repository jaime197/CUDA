﻿#include <stdio.h>
#include <stdint.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h>


typedef struct {
    uint32_t width;
    uint32_t height;
    uint16_t bpp;
    uint8_t* data;
} BMPImage;

BMPImage load_bmp(const char* filename) {
    BMPImage image;
    FILE* file = fopen(filename, "rb");
    if (!file) {
        perror("Error al abrir el archivo BMP");
        exit(EXIT_FAILURE);
    }

    fseek(file, 18, SEEK_SET);
    fread(&image.width, sizeof(uint32_t), 1, file);
    fread(&image.height, sizeof(uint32_t), 1, file);

    fseek(file, 28, SEEK_SET);
    fread(&image.bpp, sizeof(uint16_t), 1, file);

    fseek(file, 54, SEEK_SET);
    unsigned long data_size = (size_t)(image.width * image.height * (image.bpp / 8));
    image.data = (uint8_t*)malloc(data_size);
    if (!image.data) {
        fclose(file);
        perror("Error de asignación de memoria para datos de píxeles");
        exit(EXIT_FAILURE);
    }

    fread(image.data, sizeof(uint8_t), data_size, file);
    fclose(file);
    return image;
}

__global__ void convolutionKernel(uint8_t* input, uint8_t* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) {
        for (int c = 0; c < channels; ++c) {
            int pixel = 0;
            for (int ky = -1; ky <= 1; ++ky) {
                for (int kx = -1; kx <= 1; ++kx) {
                    int ix = x + kx;
                    int iy = y + ky;
                    pixel += input[(iy * width + ix) * channels + c];
                }
            }
            output[(y * width + x) * channels + c] = pixel / 9;
        }
    }
}

// Host function to call the CUDA kernel
void applyConvolutionCuda(uint8_t* input, uint8_t* output, int width, int height, int channels) {
    uint8_t* d_input, * d_output;
    size_t imageSize = width * height * channels * sizeof(uint8_t);

    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, imageSize);

    hipMemcpy(d_input, input, imageSize, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    convolutionKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height, channels);

    hipMemcpy(output, d_output, imageSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void save_bmp(const char* filename, BMPImage* image) {
    FILE* file = fopen(filename, "wb");
    if (!file) {
        perror("Error al abrir el archivo BMP para escritura");
        exit(EXIT_FAILURE);
    }

    uint32_t row_size = (image->width * (image->bpp / 8) + 3) & ~3;
    uint32_t pixel_data_size = row_size * image->height;
    uint32_t file_size = 54 + pixel_data_size;

    fwrite("BM", 1, 2, file);
    fwrite(&file_size, 4, 1, file);
    uint32_t reserved = 0;
    fwrite(&reserved, 4, 1, file);
    uint32_t offset = 54;
    fwrite(&offset, 4, 1, file);

    uint32_t dib_header_size = 40;
    fwrite(&dib_header_size, 4, 1, file);
    fwrite(&image->width, 4, 1, file);
    fwrite(&image->height, 4, 1, file);
    uint16_t planes = 1;
    fwrite(&planes, 2, 1, file);
    fwrite(&image->bpp, 2, 1, file);
    uint32_t compression = 0;
    fwrite(&compression, 4, 1, file);
    fwrite(&pixel_data_size, 4, 1, file);
    uint32_t resolution = 2835;
    fwrite(&resolution, 4, 1, file);
    fwrite(&resolution, 4, 1, file);
    uint32_t colors = 0;
    fwrite(&colors, 4, 1, file);
    fwrite(&colors, 4, 1, file);

    uint8_t padding[3] = { 0 };
    for (int y = 0; y < image->height; y++) {
        fwrite(image->data + (y * image->width * (image->bpp / 8)), 1, image->width * (image->bpp / 8), file);
        fwrite(padding, 1, (4 - (image->width * (image->bpp / 8)) % 4) % 4, file);
    }

    fclose(file);
}

int main() {
    const char* filename = "images/pikachuBMP.bmp";
    BMPImage image = load_bmp(filename);

    int width = image.width;
    int height = image.height;
    int channels = image.bpp / 8;

    uint8_t* output = (uint8_t*)malloc(width * height * channels * sizeof(uint8_t));

    applyConvolutionCuda(image.data, output, width, height, channels);

    save_bmp("images/output_cuda.bmp", &image);

    free(output);
    free(image.data);

    return 0;
}

